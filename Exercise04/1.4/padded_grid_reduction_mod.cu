#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random.h>
#include <thrust/extrema.h>
#include <cmath>
#include <iomanip>
#include <float.h>
#include <chrono>

// This example computes the minimum and maximum values
// over a padded grid.  The padded values are not considered
// during the reduction operation.


// transform a tuple (int,value) into a tuple (bool,value,value)
// where the bool is true for valid grid values and false for 
// values in the padded region of the grid
template <typename IndexType, typename ValueType>
struct transform_tuple : 
    public thrust::unary_function< thrust::tuple<IndexType,ValueType>, 
                                   thrust::tuple<bool,ValueType,ValueType> >
{
  typedef typename thrust::tuple<IndexType,ValueType>      InputTuple;
  typedef typename thrust::tuple<bool,ValueType,ValueType> OutputTuple;

  IndexType n, N;

  transform_tuple(IndexType n, IndexType N) : n(n), N(N) {}

  __host__ __device__
    OutputTuple operator()(const InputTuple& t) const
    { 
      bool is_valid = (thrust::get<0>(t) % N) < n;
      return OutputTuple(is_valid, thrust::get<1>(t), thrust::get<1>(t));
    }
};


// reduce two tuples (bool,value,value) into a single tuple such that output
// contains the smallest and largest *valid* values.
template <typename IndexType, typename ValueType>
struct reduce_tuple :
    public thrust::binary_function< thrust::tuple<bool,ValueType,ValueType>,
                                    thrust::tuple<bool,ValueType,ValueType>,
                                    thrust::tuple<bool,ValueType,ValueType> >
{
  typedef typename thrust::tuple<bool,ValueType,ValueType> Tuple;

  __host__ __device__
    Tuple operator()(const Tuple& t0, const Tuple& t1) const
    { 
      if(thrust::get<0>(t0) && thrust::get<0>(t1)) // both valid
        return Tuple(true, 
            thrust::min(thrust::get<1>(t0), thrust::get<1>(t1)),
            thrust::max(thrust::get<2>(t0), thrust::get<2>(t1)));
      else if (thrust::get<0>(t0))
        return t0;
      else if (thrust::get<0>(t1))
        return t1;
      else
        return t1; // if neither is valid then it doesn't matter what we return
    }
};

template <typename ValueType>
struct transform_tuple_modified
{
  typedef typename thrust::tuple<ValueType,ValueType> OutputTuple;
  
  transform_tuple_modified() {}
  __host__ __device__
    OutputTuple operator()(const ValueType& t) const
    {
      return OutputTuple(t, t);
    }
};

template <typename ValueType>
struct reduce_tuple_modified
{
  typedef typename thrust::tuple<ValueType,ValueType> Tuple;
  
  __host__ __device__
    Tuple operator()(const Tuple& t0, const Tuple& t1) const
    {
      return Tuple(
        thrust::min(thrust::get<0>(t0), thrust::get<0>(t1)),
        thrust::max(thrust::get<1>(t0), thrust::get<1>(t1))
      );
    } 
};

int main(void)
{
  int M = 1e3;  // number of rows
  int n = 1011;  // number of columns excluding padding
  int N = 1024;  // number of columns including padding

  thrust::default_random_engine rng(12345);
  thrust::uniform_real_distribution<float> dist(0.0f, 1.0f);

  thrust::device_vector<float> data(M * N, -1);
  thrust::device_vector<float> index(M * n);

  int id = 0;
  
  // initialize valid values in grid
  for(int i = 0; i < M; i++) {
    for(int j = 0; j < n; j++) {
      data[i * N + j] = dist(rng);
      index[id] = i * N + j;
      
      id++;
    }
  }
  
  //typedef typename thurst::permutation_iterator<Iterator, TransformIterator>  PermutIt;
  //typedef PermutIt iterator;
  
  //auto pi_start = thrust::make_permutation_iterator(data.begin(), index.end());
 // auto pi_stop  = thrust::make_permutation_iterator(data.end(), index.end());
  
  
  // print full grid
  /*
  std::cout << "padded grid" << std::endl;
  std::cout << std::fixed << std::setprecision(4);
  for(int i = 0; i < M; i++)
  {
    std::cout << " ";
    for(int j = 0; j < N; j++)
    {
      std::cout << data[i * N + j] << " ";
    }   
    std::cout << "\n";
  }
  std::cout << "\n";
  */
  
  // compute min & max over valid region of the 2d grid
  std::cout << "THRUST EXAMPLE: " << std::endl;
  std::cout << "======================== " << std::endl;
  {
    typedef thrust::tuple<bool, float, float> result_type;
    result_type                 init(true, FLT_MAX, -FLT_MAX); // initial value
    transform_tuple<int,float>  unary_op(n, N);                // transformation operator
    reduce_tuple<int,float>     binary_op;                     // reduction operator
    
    hipDeviceSynchronize();
    auto startTime = std::chrono::high_resolution_clock::now();
    
    result_type result = 
      thrust::transform_reduce(
          thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0), data.begin())),
          thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0), data.begin())) + data.size(),
          unary_op,
          init,
          binary_op);
    
    hipDeviceSynchronize();
    auto endTime = std::chrono::high_resolution_clock::now();
    std::cout << "seconds (original): " << (endTime-startTime).count() << std::endl;
    
    std::cout << "minimum value: " << thrust::get<1>(result) << std::endl;
    std::cout << "maximum value: " << thrust::get<2>(result) << std::endl;
  }
  
  std::cout << std::endl;
  
  std::cout << "THRUST EXAMPLE MODIFIED: " << std::endl;
  std::cout << "======================== " << std::endl;
  {
    typedef thrust::tuple<float, float> result_type;
  
    result_type                         init(FLT_MAX, -FLT_MAX);
    transform_tuple_modified<float>     unary_op;
    reduce_tuple_modified<float>        binary_op;
    
    hipDeviceSynchronize();
    auto startTime = std::chrono::high_resolution_clock::now();
    
    result_type result = 
      thrust::transform_reduce(
          thrust::make_permutation_iterator(data.begin(), index.begin()),
          thrust::make_permutation_iterator(data.end(), index.end()),
          unary_op,
          init,
          binary_op
      );
      
    hipDeviceSynchronize();
    auto endTime = std::chrono::high_resolution_clock::now();
    std::cout << "seconds (modified): " << (endTime-startTime).count() << std::endl;
    
    std::cout << "minimum value: " << thrust::get<0>(result) << std::endl;
    std::cout << "maximum value: " << thrust::get<1>(result) << std::endl;
  }
  
  return 0;
}

