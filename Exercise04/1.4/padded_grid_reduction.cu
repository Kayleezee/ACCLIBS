#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random.h>
#include <thrust/extrema.h>
#include <chrono>
#include <cmath>
#include <iomanip>
#include <float.h>

// This example computes the minimum and maximum values
// over a padded grid.  The padded values are not considered
// during the reduction operation.


// transform a tuple (int,value) into a tuple (bool,value,value)
// where the bool is true for valid grid values and false for 
// values in the padded region of the grid
template <typename IndexType, typename ValueType>
struct transform_tuple : 
    public thrust::unary_function< thrust::tuple<IndexType,ValueType>, 
                                   thrust::tuple<bool,ValueType,ValueType> >
{
  typedef typename thrust::tuple<IndexType,ValueType>      InputTuple;
  typedef typename thrust::tuple<bool,ValueType,ValueType> OutputTuple;

  IndexType n, N;

  transform_tuple(IndexType n, IndexType N) : n(n), N(N) {}

  __host__ __device__
    OutputTuple operator()(const InputTuple& t) const
    { 
      bool is_valid = (thrust::get<0>(t) % N) < n;
      return OutputTuple(is_valid, thrust::get<1>(t), thrust::get<1>(t));
    }
};


// reduce two tuples (bool,value,value) into a single tuple such that output
// contains the smallest and largest *valid* values.
template <typename IndexType, typename ValueType>
struct reduce_tuple :
    public thrust::binary_function< thrust::tuple<bool,ValueType,ValueType>,
                                    thrust::tuple<bool,ValueType,ValueType>,
                                    thrust::tuple<bool,ValueType,ValueType> >
{
  typedef typename thrust::tuple<bool,ValueType,ValueType> Tuple;

  __host__ __device__
    Tuple operator()(const Tuple& t0, const Tuple& t1) const
    { 
      if(thrust::get<0>(t0) && thrust::get<0>(t1)) // both valid
        return Tuple(true, 
            thrust::min(thrust::get<1>(t0), thrust::get<1>(t1)),
            thrust::max(thrust::get<2>(t0), thrust::get<2>(t1)));
      else if (thrust::get<0>(t0))
        return t0;
      else if (thrust::get<0>(t1))
        return t1;
      else
        return t1; // if neither is valid then it doesn't matter what we return
    }
};

int main(void)
{
  int M = 1e3;  // number of rows
  int n = 1011;  // number of columns excluding padding
  int N = 1024;  // number of columns including padding

  thrust::default_random_engine rng(12345);
  thrust::uniform_real_distribution<float> dist(0.0f, 1.0f);

  thrust::device_vector<float> data(M * N, -1);

  // initialize valid values in grid
  for(int i = 0; i < M; i++)
    for(int j = 0; j < n; j++)
      data[i * N + j] = dist(rng);


  // compute min & max over valid region of the 2d grid
  typedef thrust::tuple<bool, float, float> result_type;

  result_type                 init(true, FLT_MAX, -FLT_MAX); // initial value
  transform_tuple<int,float>  unary_op(n, N);                // transformation operator
  reduce_tuple<int,float>     binary_op;                     // reduction operator

  
  hipDeviceSynchronize();
  auto startTime = std::chrono::high_resolution_clock::now();
  
  result_type result = 
    thrust::transform_reduce(
        thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0), data.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0), data.begin())) + data.size(),
        unary_op,
        init,
        binary_op);
		
		
  hipDeviceSynchronize();
  auto endTime= std::chrono::high_resolution_clock::now();
  std::cout << "seconds: "<< (endTime-startTime).count() << std::endl;


  std::cout << "minimum value: " << thrust::get<1>(result) << std::endl;
  std::cout << "maximum value: " << thrust::get<2>(result) << std::endl;

  return 0;
}

